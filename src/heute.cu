#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

extern "C" {

    __global__ void apply_threshold(hipfftComplex* data, float* amplitude, int n, float threshold) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n) {
            float re = data[idx].x;
            float im = data[idx].y;
            float amp = sqrtf(re * re + im * im);
            amplitude[idx] = amp > threshold ? amp : 0.0f;
        }
    }

    extern void fft(hipfftComplex* data, float* amplitude, int n, float threshold, int blockSize) {
        // Create cuFFT plan
        hipfftHandle plan;
        hipfftPlan1d(&plan, n, HIPFFT_C2C, 1);

        // Execute FFT
        hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);

        // Launch the kernel to apply the threshold
        int numBlocks = (n + blockSize - 1) / blockSize;
        apply_threshold<<<numBlocks, blockSize>>>(data, amplitude, n, threshold);

        // Destroy cuFFT plan
        hipfftDestroy(plan);

        // Wait for GPU to finish
        hipDeviceSynchronize();
    }
}
