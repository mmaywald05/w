#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <chrono>
#include <thrust/complex.h>


using namespace std::chrono;
#define M_PI 3.14159265359

typedef struct {
    char    ChunkID[4];
    int32_t ChunkSize;
    char    Format[4];
    char    Subchunk1ID[4];
    int32_t Subchunk1Size;
    int16_t AudioFormat;
    int16_t NumChannels;
    int32_t SampleRate;
    int32_t ByteRate;
    int16_t BlockAlign;
    int16_t BitsPerSample;
    char    Subchunk2ID[4];
    int32_t Subchunk2Size;
} WavHeader;

// Define a complex number type
typedef float2 Complex;
__global__ void dftKernel(const Complex* input, Complex* output, int N, int k, int s, int numBlocks) {
    int tid = threadIdx.x;  // Index within the block (frequency bin)

    if (tid < k) {
        Complex sum = make_float2(0.0f, 0.0f);

        for (int b = 0; b < numBlocks; ++b) {
            Complex tempSum = make_float2(0.0f, 0.0f);
            for (int n = 0; n < k; ++n) {
                int index = b * s + n;
                if (index < N) {
                    float angle = 2.0f * M_PI * tid * n / k;
                    float cosAngle = cosf(angle);
                    float sinAngle = -sinf(angle);  // Note the negative sign for the DFT

                    tempSum.x += input[index].x * cosAngle - input[index].y * sinAngle;
                    tempSum.y += input[index].x * sinAngle + input[index].y * cosAngle;
                }
            }
            sum.x += tempSum.x;
            sum.y += tempSum.y;
        }
        output[tid] = make_float2(sum.x / numBlocks, sum.y / numBlocks);
    }
}

__device__ __host__ Complex make_complex(float real, float imag) {
    Complex c;
    c.x = real;
    c.y = imag;
    return c;
}

__device__ __host__ Complex complex_add(const Complex& a, const Complex& b) {
    return make_complex(a.x + b.x, a.y + b.y);
}

__device__ __host__ Complex complex_mul(const Complex& a, const Complex& b) {
    return make_complex(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
}

__device__ __host__ float complex_mag(const Complex& c) {
    return sqrtf(c.x * c.x + c.y * c.y);
}


__global__ void mydftkernel(const Complex* input, float* magnitudes, int N, int k, int s, int numBlocks){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int startIndex = tid * s;
    int endIndex = startIndex + k;

    if(tid < numBlocks){
        for(int i = startIndex; i < endIndex; ++i){
            Complex number = make_complex(0,0);
            for(int j = startIndex; j < endIndex; ++j){
                double angle = 2 * M_PI * i * j / k;
                Complex w = make_complex(cosf(angle), -sinf(angle));
                Complex prod = complex_mul(input[j], w);
                number = complex_add(number, prod);

                // das hier ist kritisch, ich glaube das geht so nicht, besser die magnituden hier ausrechnen und einfach so übergeben
            }
            float mag = complex_mag(number)/numBlocks;
             // Print the magnitude before it is added to the magnitudes array for debugging
            atomicAdd(&magnitudes[(i-startIndex)], mag);
        }
    }
}

void computeDFTBlocks(const Complex* h_input, float* h_magnitudes, int N, int k, int s) {
    int numBlocks = (N - k) / s + 1;  // Calculate the number of blocks

    Complex* d_input;

    float* d_magnitudes;



    // Allocate device memory
    hipMalloc((void**)&d_input, N * sizeof(Complex));

    hipMalloc((void**)&d_magnitudes, k * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, h_input, N * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemset(d_magnitudes, 0, k * sizeof(float));
    // Launch the DFT kernel with enough blocks and threads to cover all frequency bins



    mydftkernel<<<1024, 1024>>>(d_input, d_magnitudes, N, k, s, numBlocks);
    hipDeviceSynchronize();

    // Launch the magnitude kernel
    //magnitudeKernel<<<1, blockSize>>>(d_output, d_magnitudes, k);

    // Copy the results back to the host
    hipMemcpy(h_magnitudes, d_magnitudes, k * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_magnitudes);
}



void saveArrayToFile(const float* values, int numSamples, const std::string& filename) {
    std::ofstream outFile(filename); // Create an output file stream

    if (!outFile) {
        std::cerr << "Error: Could not open file for writing." << std::endl;
        return;
    }

    for (int i = 0; i < numSamples; ++i) {
        outFile << values[i] << std::endl; // Write each value on a new line
    }

    outFile.close(); // Close the file stream
}

std::vector<float> readWav(const std::string& filePath, int &sampleRate) {
      std::ifstream file(filePath, std::ios::binary);
      WavHeader header;

      if (!file.read(reinterpret_cast<char*>(&header), sizeof(WavHeader))) {
          throw std::runtime_error("Failed to read WAV file header.");
      }

      if (strncmp(header.ChunkID, "RIFF", 4) != 0 || strncmp(header.Format, "WAVE", 4) != 0) {
          throw std::runtime_error("File is not a valid WAV file.");
      }

      if (header.AudioFormat != 1) {
          throw std::runtime_error("Unsupported audio format. Only PCM format supported.");
      }

      if (header.BitsPerSample != 16) {
          throw std::runtime_error("Unsupported sample size. Only 16-bit samples supported.");
      }

      int sampleCount = header.Subchunk2Size / sizeof(int16_t);
      sampleRate = header.SampleRate;

    sampleCount = header.Subchunk2Size / sizeof(int16_t);
    std::vector<int16_t> buffer(sampleCount);

       file.read(reinterpret_cast<char*>(buffer.data()), header.Subchunk2Size);

      file.close();

      std::vector<float> samples(buffer.size());

    // Convert int16_t samples to float and normalize to [-1, 1]
     float maxAbsValue = 0;
     for (size_t i = 0; i < buffer.size(); ++i) {
        samples[i] = static_cast<float>(buffer[i]) / std::numeric_limits<int16_t>::max();
        if (std::abs(samples[i]) > maxAbsValue) {
            maxAbsValue = std::abs(samples[i]);
        }
     }

    if (maxAbsValue > 0) {
        for (size_t i = 0; i < sampleCount; ++i) {
            samples[i] /= maxAbsValue;
        }
    }

    return samples;
}


int main(int argc, char *argv[]) {
    auto start = high_resolution_clock::now();
    std::string filePath = argv[1];

    int sampleRate;
    std::vector<float> samples = readWav(filePath, sampleRate);

    //readWavFile(filePath, samples, sampleRate);
    int N = samples.size();;  // Number of source file samples
    int k = 512;    // blocksize
    int s = 64;     // shift
    int numBlocks = (N - k) / s + 1;


    Complex* h_input = (Complex*)malloc(N * sizeof(Complex));
    float* h_magnitudes = (float*)malloc(k * sizeof(float));

    // Initialize input data (example: sine wave)
    for (int n = 0; n < N; ++n) {
        h_input[n].x = samples[n];
        h_input[n].y = 0.0f;

    }


    std::cout << "Starting DFT...";
    // Compute the DFT
    computeDFTBlocks(h_input, h_magnitudes, N, k, s);
    std::cout << "done:" <<std::endl;

    // Print the magnitudes of the frequency bins
    std::cout << "k = Blocksize = " << k << std::endl;




    int pos, neg,zero;
    float max = 0;
    float min = FLT_MAX;
    int maxIndex = 0;
    int minIndex = 0;

    pos =0; neg=0;zero=0;
    for(int i = 0; i < k; ++i){
        h_magnitudes[i] = h_magnitudes[i] / numBlocks;

        if(h_magnitudes[i] > 0){
            ++pos;
        }else if (h_magnitudes[i] < 0){
            ++neg;
        }else{
            ++zero;
        }

        if(h_magnitudes[i]  > max){
            max = h_magnitudes[i];
            maxIndex = i;
        }
        if(h_magnitudes[i]< min ){
            minIndex = i;
            min = h_magnitudes[i];
        }


    }
    for(int i =0;  i< k; ++i){
        h_magnitudes[i] = (h_magnitudes[i]-min)/(max-min);
    }


    saveArrayToFile(h_magnitudes, k, "magnitudes.txt");

/*
    for(int i = 0; i < k;  ++i){
        h_magnitudes[i] = std::round(h_magnitudes[i]*100)/100;
    }
    */

    for(int i = 0; i < k; ++i){
        double frequency = i * sampleRate / k;
        printf("Frequency %f: Magnitude = %f\n", frequency, h_magnitudes[i]);
    }

    // Free host memory
    free(h_input);
    free(h_magnitudes);


    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    std::cout <<"GPU DFT took "<< duration.count()/1000 << "ms." << std::endl;


    return 0;
}
