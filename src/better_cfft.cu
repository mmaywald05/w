#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <chrono>
#include <thrust/complex.h>


using namespace std::chrono;
#define M_PI 3.14159265359

struct WAVHeader {
    char riff[4];        // "RIFF"
    int overall_size;    // File size minus 8 bytes
    char wave[4];        // "WAVE"
    char fmt_chunk_marker[4];  // "fmt "
    int length_of_fmt;   // Length of format data (usually 16)
    short format_type;   // Format type (1 is PCM)
    short channels;      // Number of channels
    int sample_rate;     // Sampling rate (blocks per second)
    int byterate;        // Bytes per second
    short block_align;   // 2=16-bit mono, 4=16-bit stereo
    short bits_per_sample; // Number of bits per sample
    char data_chunk_header[4]; // "data"
    int data_size;       // Size of data
};

// Define a complex number type
typedef float2 Complex;
__global__ void dftKernel(const Complex* input, Complex* output, int N, int k, int s, int numBlocks) {
    int tid = threadIdx.x;  // Index within the block (frequency bin)

    if (tid < k) {
        Complex sum = make_float2(0.0f, 0.0f);

        for (int b = 0; b < numBlocks; ++b) {
            Complex tempSum = make_float2(0.0f, 0.0f);
            for (int n = 0; n < k; ++n) {
                int index = b * s + n;
                if (index < N) {
                    float angle = 2.0f * M_PI * tid * n / k;
                    float cosAngle = cosf(angle);
                    float sinAngle = -sinf(angle);  // Note the negative sign for the DFT

                    tempSum.x += input[index].x * cosAngle - input[index].y * sinAngle;
                    tempSum.y += input[index].x * sinAngle + input[index].y * cosAngle;
                }
            }
            sum.x += tempSum.x;
            sum.y += tempSum.y;
        }

        output[tid] = make_float2(sum.x / numBlocks, sum.y / numBlocks);
    }
}

__device__ __host__ Complex make_complex(float real, float imag) {
    Complex c;
    c.x = real;
    c.y = imag;
    return c;
}

__device__ __host__ Complex complex_add(const Complex& a, const Complex& b) {
    return make_complex(a.x + b.x, a.y + b.y);
}

__device__ __host__ Complex complex_mul(const Complex& a, const Complex& b) {
    return make_complex(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
}

__device__ __host__ float complex_mag(const Complex& c) {
    return sqrtf(c.x * c.x + c.y * c.y);
}


__global__ void mydftkernel(const Complex* input, float* magnitudes, int N, int k, int s, int numBlocks){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int startIndex = tid * s;
    int endIndex = startIndex + k;

    __syncthreads();

    if(tid < numBlocks){
        for(int i = startIndex; i < endIndex; ++i){
            Complex number = make_complex(0,0);
            for(int j = startIndex; j < endIndex; ++j){
                double angle = 2 * M_PI * i * j / k;
                Complex w = make_complex(cosf(angle), -sinf(angle));
                Complex prod = complex_mul(input[j], w);
                number = complex_add(number, prod);

                // das hier ist kritisch, ich glaube das geht so nicht, besser die magnituden hier ausrechnen und einfach so übergeben
            }
            float mag = complex_mag(number)/numBlocks;
             // Print the magnitude before it is added to the magnitudes array for debugging
            atomicAdd(&magnitudes[(i-startIndex)], mag);
        }
    }
}

void computeDFTBlocks(const Complex* h_input, float* h_magnitudes, int N, int k, int s) {
    int numBlocks = (N - k) / s + 1;  // Calculate the number of blocks

    Complex* d_input;

    float* d_magnitudes;



    // Allocate device memory
    hipMalloc((void**)&d_input, N * sizeof(Complex));

    hipMalloc((void**)&d_magnitudes, k * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, h_input, N * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemset(d_magnitudes, 0, k * sizeof(float));
    // Launch the DFT kernel with enough blocks and threads to cover all frequency bins



    mydftkernel<<<1024, 1024>>>(d_input, d_magnitudes, N, k, s, numBlocks);
    hipDeviceSynchronize();

    // Launch the magnitude kernel
    //magnitudeKernel<<<1, blockSize>>>(d_output, d_magnitudes, k);

    // Copy the results back to the host
    hipMemcpy(h_magnitudes, d_magnitudes, k * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_magnitudes);
}



void readWavFile(const std::string &filePath, std::vector<float> &samples, int &sampleRate) {
    std::ifstream file(filePath, std::ios::binary);
    if (!file) {
        std::cerr << "Error opening file: " << filePath << std::endl;
        exit(1);
    }

    WAVHeader header;
    file.read(reinterpret_cast<char*>(&header), sizeof(WAVHeader));

    if (std::strncmp(header.riff, "RIFF", 4) != 0 || std::strncmp(header.wave, "WAVE", 4) != 0) {
        std::cerr << "Invalid WAV file format" << std::endl;
        exit(1);
    }

    sampleRate = header.sample_rate;
    int numSamples = header.data_size / (header.bits_per_sample / 8);
    samples.resize(numSamples);

    if (header.bits_per_sample == 16) {
        std::vector<short> tempSamples(numSamples);
        file.read(reinterpret_cast<char*>(tempSamples.data()), header.data_size);
        for (int i = 0; i < numSamples; ++i) {
            samples[i] = tempSamples[i] / 32768.0f;
        }
    } else if (header.bits_per_sample == 8) {
        std::vector<unsigned char> tempSamples(numSamples);
        file.read(reinterpret_cast<char*>(tempSamples.data()), header.data_size);
        for (int i = 0; i < numSamples; ++i) {
            samples[i] = tempSamples[i] / 128.0f - 1.0f;
        }
    } else {
        std::cerr << "Unsupported bit depth: " << header.bits_per_sample << std::endl;
        exit(1);
    }
}

void plotHistogram(const float* values, int numSamples, int height) {
    if (numSamples <= 0 || height <= 0) {
        std::cerr << "Number of samples and height must be positive integers." << std::endl;
        return;
    }

    // Calculate the maximum value in the array for scaling
    float maxValue = 0.0f;
    for (int i = 0; i < numSamples; ++i) {
        if (values[i] > maxValue) {
            maxValue = values[i];
        }
    }

    // Draw histogram from top to bottom
    for (int row = height - 1; row >= 0; --row) {
        float threshold = (static_cast<float>(row) / height) * maxValue;

        for (int col = 0; col < numSamples; ++col) {
            if (values[col] >= threshold) {
                std::cout << "*";
            } else {
                std::cout << ".";
            }
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

void saveArrayToFile(const float* values, int numSamples, const std::string& filename) {
    std::ofstream outFile(filename); // Create an output file stream

    if (!outFile) {
        std::cerr << "Error: Could not open file for writing." << std::endl;
        return;
    }

    for (int i = 0; i < numSamples; ++i) {
        outFile << values[i] << std::endl; // Write each value on a new line
    }

    outFile.close(); // Close the file stream
}

int main(int argc, char *argv[]) {
    auto start = high_resolution_clock::now();
    std::string filePath = argv[1];
    std::vector<float> samples;
    int sampleRate;
    readWavFile(filePath, samples, sampleRate);

    int N = samples.size();;  // Number of source file samples
    int k = 512;    // blocksize
    int s = 64;     // shift
    int numBlocks = (N - k) / s + 1;


    Complex* h_input = (Complex*)malloc(N * sizeof(Complex));
    float* h_magnitudes = (float*)malloc(k * sizeof(float));

    // Initialize input data (example: sine wave)
    for (int n = 0; n < N; ++n) {
        h_input[n].x = samples[n];
        h_input[n].y = 0.0f;

    }

    for(int i = 0; i < 50 ; ++i){
        std::cout << h_input[i].x << std::endl;
    }




    std::cout << "Starting DFT...";
    // Compute the DFT
    computeDFTBlocks(h_input, h_magnitudes, N, k, s);
    std::cout << "done:" <<std::endl;

    // Print the magnitudes of the frequency bins
    std::cout << "k = Blocksize = " << k << std::endl;




    int pos, neg,zero;
    float max = 0;
    float min = FLT_MAX;
    int maxIndex = 0;
    int minIndex = 0;

    pos =0; neg=0;zero=0;
    for(int i = 0; i < k; ++i){
        if(h_magnitudes[i] > 0){
            ++pos;
        }else if (h_magnitudes[i] < 0){
            ++neg;
        }else{
            ++zero;
        }

        if(h_magnitudes[i]  > max){
            max = h_magnitudes[i];
            maxIndex = i;
        }
        if(h_magnitudes[i]< min ){
            minIndex = i;
            min = h_magnitudes[i];
        }


    }
    for(int i =0;  i< k; ++i){
        h_magnitudes[i] = (h_magnitudes[i]-min)/(max-min);
    }
    for(int i =0; i < k ; ++i){

        printf("Frequency bin %d: Magnitude = %f\n", i, h_magnitudes[i]);
    }
    std::cout << "positive: " << pos << std::endl;
    std::cout << "negative: " << neg << std::endl;
    std::cout << "zero: " << zero << std::endl;

    std::cout << "max: " << maxIndex << "->" << max <<  " | min: "<< minIndex << "->"  << min << std::endl;

    //plotHistogram(h_magnitudes, k, 10);

    saveArrayToFile(h_magnitudes, k, "magnitudes.txt");

    for(int i = 0; i < k; ++i){
        double frequency = i * sampleRate / k;
        printf("Frequency %f: Magnitude = %f\n", frequency, h_magnitudes[i]);
    }


    /*


     System.out.println("Frequenz (Hz)\tAmplitudenmittelwert");
            for (int i = 0; i < amplitudeSums.length; i++) {
                double averageAmplitude = amplitudeSums[i] / numBlocks;
                if (averageAmplitude > threshold) {
                    double frequency = (double) i * sampleRate / blockSize;
                    System.out.printf("%.2f\t\t%.5f%n", frequency, averageAmplitude);
                }
            }
    for (int k = 0; k < N; ++k) {
        printf("Frequency bin %d: Magnitude = %f\n", k, h_magnitudes[k]);
    }
    */


    // Free host memory
    free(h_input);
    free(h_magnitudes);


    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    std::cout <<"CUDA FFT took "<< duration.count()/1000 << "ms." << std::endl;


    return 0;
}
